#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

#define THREADS_PER_BLOCK 1024

inline int GET_BLOCKS(const int N) {
  int optimal_block_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  int max_block_num = 65000;
  return min(optimal_block_num, max_block_num);
}

template <typename scalar_t>
__global__ void MaskedIm2colForward(const int n, const scalar_t *data_im,
                                    const int height, const int width,
                                    const int kernel_h, const int kernel_w,
                                    const int pad_h, const int pad_w,
                                    const long *mask_h_idx,
                                    const long *mask_w_idx, const int mask_cnt,
                                    scalar_t *data_col) {
  // mask_cnt * channels
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int m_index = index % mask_cnt;
    const int h_col = mask_h_idx[m_index];
    const int w_col = mask_w_idx[m_index];
    const int c_im = index / mask_cnt;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col - pad_h;
    const int w_offset = w_col - pad_w;
    scalar_t *data_col_ptr = data_col + c_col * mask_cnt + m_index;
    for (int i = 0; i < kernel_h; ++i) {
      int h_im = h_offset + i;
      for (int j = 0; j < kernel_w; ++j) {
        int w_im = w_offset + j;
        if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
          *data_col_ptr =
              (scalar_t)data_im[(c_im * height + h_im) * width + w_im];
        } else {
          *data_col_ptr = 0.0;
        }
        data_col_ptr += mask_cnt;
      }
    }
  }
}

int MaskedIm2colForwardLaucher(const at::Tensor bottom_data, const int height,
                               const int width, const int channels,
                               const int kernel_h, const int kernel_w,
                               const int pad_h, const int pad_w,
                               const at::Tensor mask_h_idx,
                               const at::Tensor mask_w_idx, const int mask_cnt,
                               at::Tensor top_data) {
  const int output_size = mask_cnt * channels;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      bottom_data.type(), "MaskedIm2colLaucherForward", ([&] {
        const scalar_t *bottom_data_ = bottom_data.data<scalar_t>();
        const long *mask_h_idx_ = mask_h_idx.data<long>();
        const long *mask_w_idx_ = mask_w_idx.data<long>();
        scalar_t *top_data_ = top_data.data<scalar_t>();
        MaskedIm2colForward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, bottom_data_, height, width, kernel_h, kernel_w,
                pad_h, pad_w, mask_h_idx_, mask_w_idx_, mask_cnt, top_data_);
      }));
  TORCH_CHECK(hipGetLastError() == hipSuccess);
  return 1;
}

template <typename scalar_t>
__global__ void MaskedCol2imForward(const int n, const scalar_t *data_col,
                                    const int height, const int width,
                                    const int channels, const long *mask_h_idx,
                                    const long *mask_w_idx, const int mask_cnt,
                                    scalar_t *data_im) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int m_index = index % mask_cnt;
    const int h_im = mask_h_idx[m_index];
    const int w_im = mask_w_idx[m_index];
    const int c_im = index / mask_cnt;
    // int kernel_extent_w = (kernel_w - 1) + 1;
    // int kernel_extent_h = (kernel_h - 1) + 1;
    // compute the start and end of the output
    data_im[(c_im * height + h_im) * width + w_im] = data_col[index];
  }
}

int MaskedCol2imForwardLaucher(const at::Tensor bottom_data, const int height,
                               const int width, const int channels,
                               const at::Tensor mask_h_idx,
                               const at::Tensor mask_w_idx, const int mask_cnt,
                               at::Tensor top_data) {
  const int output_size = mask_cnt * channels;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      bottom_data.type(), "MaskedCol2imLaucherForward", ([&] {
        const scalar_t *bottom_data_ = bottom_data.data<scalar_t>();
        const long *mask_h_idx_ = mask_h_idx.data<long>();
        const long *mask_w_idx_ = mask_w_idx.data<long>();
        scalar_t *top_data_ = top_data.data<scalar_t>();

        MaskedCol2imForward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, bottom_data_, height, width, channels, mask_h_idx_,
                mask_w_idx_, mask_cnt, top_data_);
      }));
  TORCH_CHECK(hipGetLastError() == hipSuccess);
  return 1;
}
